#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include <vector>
#include <stdio.h>
#include <type_traits>
#include <cuda/barrier>

using namespace torch::indexing;

template<typename N, int64_t d>
using PAcc64 = torch::PackedTensorAccessor64<N,d,torch::RestrictPtrTraits>;

__inline__ int iceil(int in, int div){ return (in + div - 1)/div;}						  
		  
#define POPC1B(arr) (__popc(arr[0]) << 0 )| \
					(__popc(arr[1]) << 8 )| \
					(__popc(arr[2]) << 16)| \
					(__popc(arr[3]) << 24);	
					
long lowestPowerof2(unsigned x)
{
	x--;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
	x += 1;
    return (x == 0) ? 1 : x;
}

template <unsigned int blockSize>
__device__ __inline__ void h_warpReduce(volatile int *sdata, unsigned int tid) {
	if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
	if (blockSize >= 32) sdata[tid] += sdata[tid + 16]; 
	if (blockSize >= 16) sdata[tid] += sdata[tid +  8];
	if (blockSize >=  8) sdata[tid] += sdata[tid +  4];
	if (blockSize >=  4) sdata[tid] += sdata[tid +  2];
	if (blockSize >=  2) sdata[tid] += sdata[tid +  1];
}

template<bool packed,typename inp_t>
struct load {};

template <typename inp_t>
struct load<true,inp_t> {
	__inline__ __device__ int operator()(const PAcc64<inp_t,2> inp, const int gridSize, const int row, int col) {
		int sum_count = 0;
		int d = 0, out = 0;
		//8-bit summation
		while(col < inp.size(1) - 3) {
			if(sum_count == 7) {
				out += (0x000000FF & d) + (0x000000FF & (d>>8)) + (0x000000FF & (d>>16)) + (0x000000FF & (d>>24));
				d = 0;
				sum_count = 0;
			}
			d += POPC1B((&inp[row][col])); 
			sum_count++;
			col += gridSize;
		} 
		out += (0x000000FF & d) + (0x000000FF & (d>>8)) + (0x000000FF & (d>>16)) + (0x000000FF & (d>>24));
		//Leftover summation
		while(col < inp.size(1)) {
			out += __popc(inp[row][col]);
			col += 1;
		}
		return out;
	}
};

template <typename inp_t>
struct load<false,inp_t> {
	__inline__ __device__ int operator()(const PAcc64<inp_t,2> inp, const int gridSize, const int row, int col) {
		int sum_count = 0;
		int d = 0, out = 0;
		//8-bit summation
		int s = inp.size(1);
		while(col < s) {
			if(sum_count == 7) {
				out += (0x000000FF & d) + (0x000000FF & (d>>8)) + (0x000000FF & (d>>16)) + (0x000000FF & (d>>24));
				d = 0;
				sum_count = 0;
			}
			d += inp[row][col]; 
			sum_count++;
			col += gridSize;
		} 
		out += (0x000000FF & d) + (0x000000FF & (d>>8)) + (0x000000FF & (d>>16)) + (0x000000FF & (d>>24));
		return out;
	}
};
					
template <typename scalar_t, int blockSize, int batchSize, bool packed, typename inp_t>
__global__ void h_reduce(const PAcc64<inp_t,2> inp, PAcc64<scalar_t,2> accum) {
	__shared__ int sdata[1024];
	
	const int tid = threadIdx.x;
	
	int col = (tid%blockSize)*((packed) ? 4 : 1);
	const int row = blockIdx.z*(65535*batchSize) + blockIdx.y*batchSize + tid/blockSize;
	
	const unsigned int gridSize = blockSize*gridDim.x * ((packed) ? 4 : 1);

	sdata[tid] = (row < inp.size(0) && col < inp.size(1)) ? load<packed,inp_t>()(inp, gridSize, row, col) : 0;
	
	__syncthreads();
	
	
	const int last_warp_threads = (blockSize > 64) ? 32 : blockSize/2;
	const int block_tid = tid % blockSize;
	if (blockSize >= 512) { if (block_tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (block_tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (block_tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (block_tid < last_warp_threads) h_warpReduce<blockSize>(sdata, tid);
	if (block_tid == 0 && row < accum.size(0)) accum[row][blockIdx.x] = sdata[tid];
}

template<int blockSize, typename inp_t, bool packed>
void hcount_dispatch(const torch::Tensor inp, torch::Tensor accum)
{
	const int samples_per_block = 1024/blockSize;
	auto flattened_inp   = inp.view({-1,inp.size(inp.dim()-1)});
	auto flattened_accum = accum.view({-1,accum.size(accum.dim()-1)});
	const int y_blocks = std::min(65535, iceil(flattened_inp.size(0), samples_per_block));
	const int z_blocks = iceil(iceil(flattened_inp.size(0), samples_per_block),65535);
	if(z_blocks > 65535) {printf("hcount: input too large\n"); return;}
	dim3 blocks(1,y_blocks,z_blocks);
	AT_DISPATCH_INTEGRAL_TYPES(accum.type(), "vcount_cuda", ([&] {
	h_reduce<scalar_t,blockSize,1024/blockSize, packed, inp_t><<<blocks,1024>>>(
		flattened_inp.packed_accessor64<inp_t,2,torch::RestrictPtrTraits>(),
		flattened_accum.packed_accessor64<scalar_t,2,torch::RestrictPtrTraits>());
	}));
}

template<typename inp_t, bool packed>
void call_hcount(const torch::Tensor inp, torch::Tensor accum, int blockSize) { 
	switch(blockSize) {
		case 1024: hcount_dispatch<1024, inp_t, packed> (inp, accum); break;
		case 512:  hcount_dispatch<512,  inp_t, packed> (inp, accum); break;
		case 256:  hcount_dispatch<256,  inp_t, packed> (inp, accum); break;
		case 128:  hcount_dispatch<128,  inp_t, packed> (inp, accum); break;
		case 64:   hcount_dispatch<64,   inp_t, packed> (inp, accum); break;
		case 32:   hcount_dispatch<32,   inp_t, packed> (inp, accum); break;
		case 16:   hcount_dispatch<16,   inp_t, packed> (inp, accum); break;
		case 8:    hcount_dispatch<8,    inp_t, packed> (inp, accum); break;
		case 4:    hcount_dispatch<4,    inp_t, packed> (inp, accum); break;
		case 2:    hcount_dispatch<2,    inp_t, packed> (inp, accum); break;
		case 1:    hcount_dispatch<1,    inp_t, packed> (inp, accum); break;
		default:   printf("Non power of 2 reduce requested\n");       break;
	}

}

torch::Tensor hcount_cuda(torch::Tensor inp, torch::Tensor accum) {
	
	const int max_block_size = std::min((long)1024,lowestPowerof2(inp.size(inp.dim()-1)>>3));
	call_hcount<int32_t, true>(inp, accum, max_block_size);
    return accum.index({Ellipsis,0});
}

__global__ void transpose_packed(const PAcc64<int32_t,3> inp, PAcc64<int32_t,3> trans) {
	
	__shared__ int sdata[128][4];
	
	const int tid = threadIdx.x;
	const int tile_x = blockIdx.x;
	const int tile_y = blockIdx.y;
	
	const int val_y = tid;
	const int batch = blockIdx.z;
	
	int in_x = tile_x*4;
	const int in_y = tile_y*128 + val_y;
	if(in_y < inp.size(1)) {
		for(int i = 0; i < 4; i++, in_x++) {
			sdata[val_y][i] = (in_x < inp.size(2)) ? inp[batch][in_y][in_x] : 0;
		}
	} else {
		sdata[val_y][0] = 0;
		sdata[val_y][1] = 0;
		sdata[val_y][2] = 0;
		sdata[val_y][3] = 0;
	}
	__syncthreads();
	
	const int warp_y = tid%128;
	const int transposed_x = warp_y/32;
	const int d[4] = {sdata[warp_y][0],sdata[warp_y][1],sdata[warp_y][2],sdata[warp_y][3]};
	__syncthreads();
	for(int i = 0; i < 4; i++) {
		sdata[i*32 + 0 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000000000001);
		sdata[i*32 + 1 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000000000010);
		sdata[i*32 + 2 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000000000100);
		sdata[i*32 + 3 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000000001000);
		sdata[i*32 + 4 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000000010000);
		sdata[i*32 + 5 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000000100000);
		sdata[i*32 + 6 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000001000000);
		sdata[i*32 + 7 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000010000000);
		sdata[i*32 + 8 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000000100000000);
		sdata[i*32 + 9 ][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000001000000000);
		sdata[i*32 + 10][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000010000000000);
		sdata[i*32 + 11][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000000100000000000);
		sdata[i*32 + 12][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000001000000000000);
		sdata[i*32 + 13][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000010000000000000);
		sdata[i*32 + 14][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000000100000000000000);
		sdata[i*32 + 15][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000001000000000000000);
		sdata[i*32 + 16][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000010000000000000000);
		sdata[i*32 + 17][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000000100000000000000000);
		sdata[i*32 + 18][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000001000000000000000000);
		sdata[i*32 + 19][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000010000000000000000000);
		sdata[i*32 + 20][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000000100000000000000000000);
		sdata[i*32 + 21][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000001000000000000000000000);
		sdata[i*32 + 22][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000010000000000000000000000);
		sdata[i*32 + 23][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000000100000000000000000000000);
		sdata[i*32 + 24][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000001000000000000000000000000);
		sdata[i*32 + 25][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000010000000000000000000000000);
		sdata[i*32 + 26][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00000100000000000000000000000000);
		sdata[i*32 + 27][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00001000000000000000000000000000);
		sdata[i*32 + 28][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00010000000000000000000000000000);
		sdata[i*32 + 29][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b00100000000000000000000000000000);
		sdata[i*32 + 30][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b01000000000000000000000000000000);
		sdata[i*32 + 31][transposed_x] = __ballot_sync(0xffffffff,d[i] & 0b10000000000000000000000000000000);
	}
	__syncthreads();
	
	int out_x = tile_y;
	int out_y = tile_x*128 + val_y;
	if(out_y < trans.size(1) && out_x < trans.size(2))
		trans[batch][out_y][out_x] = POPC1B(sdata[val_y]);
}



void call_bit_transpose(torch::Tensor inp, torch::Tensor trans)
{
	int threads = 128;
	const dim3 blocks((inp.size(2) + 4 - 1) / 4, (inp.size(1)+128 - 1)/128, inp.size(0));
	AT_DISPATCH_INTEGRAL_TYPES(trans.type(), "vcount_cuda", ([&] {
	transpose_packed<<<blocks,threads>>>(
		inp.packed_accessor64<int32_t,3,torch::RestrictPtrTraits>(),
		trans.packed_accessor64<int32_t,3,torch::RestrictPtrTraits>());
	}));
}


torch::Tensor vcount_cuda(torch::Tensor inp, int width) {
    auto flattened_inp   = inp.view({-1,inp.size(inp.dim()-2), inp.size(inp.dim()-1)});
    std::vector<int64_t> transpose_size(inp.sizes().data(), inp.sizes().data()+inp.dim());
    transpose_size[inp.dim()-1] = iceil(iceil(flattened_inp.size(1),32),4);
    transpose_size[inp.dim()-2] = width;
    
	torch::Tensor transposed_array = torch::zeros(transpose_size, torch::dtype(torch::kInt32).device(torch::kCUDA,0));
	auto flattened_trans = transposed_array.view({-1,transposed_array.size(transposed_array.dim()-2), transposed_array.size(transposed_array.dim()-1)});
	call_bit_transpose(flattened_inp, flattened_trans);
	int max_block_size = std::min((long)1024,lowestPowerof2(flattened_trans.size(2)>>3));
	call_hcount<int32_t, false>(flattened_trans, flattened_trans, max_block_size);
    return transposed_array.index({Ellipsis,0});
}

template <int offset>
__device__ __inline__ uint32_t extract_bits(uint32_t inp) {
	switch(inp) {
		case 0:	       return (uint32_t)0b0000<<offset;
		case 1:	       return (uint32_t)0b0001<<offset;
		case 256:	   return (uint32_t)0b0010<<offset;
		case 257:	   return (uint32_t)0b0011<<offset;
		case 65536:	   return (uint32_t)0b0100<<offset;
		case 65537:	   return (uint32_t)0b0101<<offset;
		case 65792:	   return (uint32_t)0b0110<<offset;
		case 65793:	   return (uint32_t)0b0111<<offset;
		case 16777216: return (uint32_t)0b1000<<offset;
		case 16777217: return (uint32_t)0b1001<<offset;
		case 16777472: return (uint32_t)0b1010<<offset;
		case 16777473: return (uint32_t)0b1011<<offset;
		case 16842752: return (uint32_t)0b1100<<offset;
		case 16842753: return (uint32_t)0b1101<<offset;
		case 16843008: return (uint32_t)0b1110<<offset;
		case 16843009: return (uint32_t)0b1111<<offset;
		default: __builtin_unreachable();
	}
	__builtin_unreachable();
}

template <typename scalar_t>
__global__ void pack_kernel(const PAcc64<scalar_t,2> inp, PAcc64<int32_t,2> out) {
	extern __shared__ int sdata[];
	
	const int tid = threadIdx.x;
	
	const int col_out = blockIdx.x * blockDim.x + tid; 
	const int col_in = col_out*32;
	const int i = 65535*blockIdx.z + blockIdx.y;
	
	if(col_out < out.size(1) && i < out.size(0)) {
		if(col_in + 4  <= inp.size(1)) sdata[tid]  = extract_bits<0 >(*(uint32_t*)&inp[i][col_in+0 ]);
		if(col_in + 8  <= inp.size(1)) sdata[tid] += extract_bits<4 >(*(uint32_t*)&inp[i][col_in+4 ]);
		if(col_in + 12 <= inp.size(1)) sdata[tid] += extract_bits<8 >(*(uint32_t*)&inp[i][col_in+8 ]);
		if(col_in + 16 <= inp.size(1)) sdata[tid] += extract_bits<12>(*(uint32_t*)&inp[i][col_in+12]);
		if(col_in + 20 <= inp.size(1)) sdata[tid] += extract_bits<16>(*(uint32_t*)&inp[i][col_in+16]);
		if(col_in + 24 <= inp.size(1)) sdata[tid] += extract_bits<20>(*(uint32_t*)&inp[i][col_in+20]);
		if(col_in + 28 <= inp.size(1)) sdata[tid] += extract_bits<24>(*(uint32_t*)&inp[i][col_in+24]);
		if(col_in + 32 <= inp.size(1)) sdata[tid] += extract_bits<28>(*(uint32_t*)&inp[i][col_in+28]);
		out[i][col_out] = sdata[tid];
	}
}


void pack_cuda(torch::Tensor inp, torch::Tensor out) {
	
	const int threads = 256; 
    auto flattened_inp   = inp.view({-1,inp.size(inp.dim()-1)});
	auto flattened_out = out.view({-1,out.size(out.dim()-1)});
	const int z_blocks = (flattened_out.size(0) + 65535 - 1)/65535;
	const int y_blocks = std::min((int64_t)65535,flattened_out.size(0));
	const dim3 blocks(iceil(flattened_out.size(1), threads), y_blocks, z_blocks);
	//printf("x: %d, y: %d, z: %d\n",blocks.x, blocks.y, blocks.z);
	AT_DISPATCH_INTEGRAL_TYPES_AND(torch::kBool,inp.type(), "pack_cuda", ([&] {
		pack_kernel<scalar_t><<<blocks,threads,threads*sizeof(int)>>>(
			flattened_inp.packed_accessor64<scalar_t,2,torch::RestrictPtrTraits>(),
			flattened_out.packed_accessor64<int32_t,2,torch::RestrictPtrTraits>());
	}));
	
}

template <typename scalar_t>
__global__ void unpack_kernel(const PAcc64<int32_t,2> inp, PAcc64<scalar_t,2> out) {
									  
	const int col_out = blockIdx.x * blockDim.x + threadIdx.x;
	const int i = 65535*blockIdx.z + blockIdx.y;

	const int bit_loc = col_out % 32;
	if(col_out < out.size(1)) {
		out[i][col_out] =  (((uint32_t)inp[i][col_out/warpSize]) & (1<<bit_loc))>>bit_loc;
	}
}

void unpack_cuda(torch::Tensor inp, torch::Tensor out) {
	
	const int threads = 1024; 
    auto flattened_inp   = inp.view({-1,inp.size(inp.dim()-1)});
	auto flattened_out = out.view({-1,out.size(out.dim()-1)});
	const int z_blocks = (flattened_out.size(0) + 65535 - 1)/65535;
	const int y_blocks = std::min((int64_t)65535,flattened_out.size(0));
	const dim3 blocks(iceil(flattened_out.size(1), threads),y_blocks,z_blocks);
	AT_DISPATCH_INTEGRAL_TYPES_AND(torch::kBool,out.scalar_type(), "unpack_cuda", ([&] {
		unpack_kernel<scalar_t><<<blocks,threads>>>(
			flattened_inp.packed_accessor64<int32_t,2,torch::RestrictPtrTraits>(),
			flattened_out.packed_accessor64<scalar_t,2,torch::RestrictPtrTraits>());
	}));
	
}